#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define MASK_WIDTH 3
#define O_TILE_WIDTH 12 
#define BLOCK_WIDTH (O_TILE_WIDTH + MASK_WIDTH - 1)

float* readPGM(const char* filename, int* width, int* height) {
    FILE* f = fopen(filename, "rb");
    if (f == NULL) {
        perror("Error opening file");
        return NULL;
    }

    char type[3];
    fscanf(f, "%s", type);
    if (type[0] != 'P' || type[1] != '5') {
        fprintf(stderr, "Not a PGM file\n");
        fclose(f);
        return NULL;
    }

    fscanf(f, "%d %d", width, height);
    int maxVal;
    fscanf(f, "%d", &maxVal);

    int imageSize = (*width) * (*height);
    float* imageData = (float*)malloc(imageSize * sizeof(float));

    for (int i = 0; i < imageSize; i++) {
        unsigned char pixel;
        fread(&pixel, sizeof(unsigned char), 1, f);
        imageData[i] = (float)pixel;
    }

    fclose(f);
    return imageData;
}

void writePGM(const char* filename, float* imageData, int width, int height) {
    FILE* f = fopen(filename, "wb");
    if (f == NULL) {
        perror("Error opening file");
        return;
    }

    fprintf(f, "P5\n%d %d\n255\n", width, height);

    for (int i = 0; i < width * height; i++) {
        unsigned char pixel = (unsigned char)imageData[i];
        fwrite(&pixel, sizeof(unsigned char), 1, f);
    }

    fclose(f);
}

__global__ void convolution2D(float *N, float *M, float *P, int Width) {
    __shared__ float N_ds[BLOCK_WIDTH][BLOCK_WIDTH];
    __shared__ float M_ds[MASK_WIDTH][MASK_WIDTH];

    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int row_o = blockIdx.y * O_TILE_WIDTH + ty;
    int col_o = blockIdx.x * O_TILE_WIDTH + tx;

    int row_i = row_o - MASK_WIDTH / 2;
    int col_i = col_o - MASK_WIDTH / 2;

    if ((row_i >= 0) && (row_i < Width) && (col_i >= 0) && (col_i < Width)) {
        N_ds[ty][tx] = N[row_i * Width + col_i];
    } else {
        N_ds[ty][tx] = 0.0f;
    }

    if (ty < MASK_WIDTH && tx < MASK_WIDTH) {
        M_ds[ty][tx] = M[ty * MASK_WIDTH + tx];
    }

    __syncthreads();

    float output = 0.0f;
    if (ty < O_TILE_WIDTH && tx < O_TILE_WIDTH) {
        for (int i = 0; i < MASK_WIDTH; i++) {
            for (int j = 0; j < MASK_WIDTH; j++) {
                output += M_ds[i][j] * N_ds[i + ty][j + tx];
            }
        }
        if (row_o < Width && col_o < Width) {
            P[row_o * Width + col_o] = output;
        }
    }
}

int main() {


    const char* imagePath = "input_512_512.pgm";
    int imageWidth, imageHeight;

    float* image = readPGM(imagePath, &imageWidth, &imageHeight);
    if (image == NULL) {
        return -1;
    }

    int imageSize = imageWidth * imageHeight;
    float *mask, *output;
    float *d_image, *d_mask, *d_output;

    mask = (float*)malloc(MASK_WIDTH * MASK_WIDTH * sizeof(float));
    output = (float*)malloc(imageSize * sizeof(float));

    for (int i = 0; i < MASK_WIDTH * MASK_WIDTH; i++) {
        mask[i] = 1.0f / (MASK_WIDTH * MASK_WIDTH);
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipMalloc(&d_image, imageSize * sizeof(float));
    hipMalloc(&d_mask, MASK_WIDTH * MASK_WIDTH * sizeof(float));
    hipMalloc(&d_output, imageSize * sizeof(float));

    hipMemcpy(d_image, image, imageSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_mask, mask, MASK_WIDTH * MASK_WIDTH * sizeof(float), hipMemcpyHostToDevice);

    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);
    dim3 dimGrid((imageWidth - 1) / O_TILE_WIDTH + 1, (imageHeight - 1) / O_TILE_WIDTH + 1);
    
    hipEventRecord(start);
    convolution2D<<<dimGrid, dimBlock>>>(d_image, d_mask, d_output, imageWidth);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Execution time: %f seconds\n", milliseconds/1000);
    
    hipMemcpy(output, d_output, imageSize * sizeof(float), hipMemcpyDeviceToHost);

    writePGM("output_image_GPU.pgm", output, imageWidth, imageHeight);

    hipFree(d_image);
    hipFree(d_mask);
    hipFree(d_output);

    free(image);
    free(mask);
    free(output);

    return 0;
}
